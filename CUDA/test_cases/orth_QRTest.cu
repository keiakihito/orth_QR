// includes, system
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

/*Using updated (v2) interfaces to cublas*/
#include<hipblas.h>
#include<hip/hip_runtime.h>
#include<hipsparse.h>
#include <hipsolver.h>
#include<sys/time.h>


//Utilities
// helper function CUDA error checking and initialization 
#include "../include/functions/orth_QR.h"



void orth_QRtest1();
void orth_QRtest2();
void orth_QRtest3();
void orth_QRtest4();
void orth_QRtest5();



int main(int argc, char** argv)
{
    printf("\n\n~~orth_Test()~~\n\n");

    // printf("\n\n🔍🔍🔍Test case 1🔍🔍🔍\n");
    // orth_QRtest1();
    
    // printf("\n\n🔍🔍🔍Test case 2🔍🔍🔍\n");
    // orth_QRtest2();

    // printf("\n\n🔍🔍🔍Test case 3🔍🔍🔍\n");
    // orth_QRtest3();

    // printf("\n\n🔍🔍🔍Test case 4🔍🔍🔍\n");
    // orth_QRtest4();

    printf("\n\n🔍🔍🔍Test case 5🔍🔍🔍\n");
    orth_QRtest5();

    printf("\n= = = End of orth_test  = = = \n\n");


}// end of main


void orth_QRtest1()
{
       /*
    Z = | 1.0  5.0  9.0 |
        | 2.0  6.0  10.0|
        | 3.0  7.0  11.0|
        | 4.0  8.0  12.0| 
    */

    // Define the dense matrixB column major
    double mtxZ[] = {
    1.0, 2.0, 3.0, 4.0,
    5.0, 6.0, 7.0, 8.0,
    9.0, 10.0, 11.0, 12.0,
    };

    int numOfRow = 4;
    int numOfClm = 3;
    int crntRank = 3;

    double* mtxZ_d = NULL;
    double* mtxQ_trnc_d = NULL;

    bool debug = true;


    CHECK(hipMalloc((void**)&mtxZ_d, numOfRow * numOfClm * sizeof(double)));
    CHECK(hipMemcpy(mtxZ_d, mtxZ, numOfRow * numOfClm * sizeof(double), hipMemcpyHostToDevice));

    if(debug){
        printf("\n\n~~mtxZ~~\n\n");
        print_mtx_clm_d(mtxZ_d, numOfRow, numOfClm);
    }


    orth_QR(&mtxQ_trnc_d, mtxZ_d, numOfRow, numOfClm, crntRank);


    // if(mtxQ_trnc_d != NULL){
    //     printf("\n\n~~mtxQ_trnc_d~~\n\n");
    //     print_mtx_clm_d(mtxQ_trnc_d, numOfRow, crntRank);
    // }
    
    printf("\n\n~~Current Rarnk = %d~~\n\n", crntRank);


} // end of orth_QRtest1()

void orth_QRtest2()
{
 // Define the dense matrixB column major
    double mtxZ[] = {
    1.1, 0.8, 3.0, 2.2, 0.2, 0.7,
    2.2, 1.6, 4.1, 3.3, 0.3, 0.8,
    3.3, 2.4, 5.2, 4.4, 0.4, 1.1,
    4.4, 3.2, 6.3, 5.5, 0.5, 1.5,
    5.5, 2.3, 0.7, 1.7, 0.6, 3.2
    };

    int numOfRow = 6;
    int numOfClm = 5;
    int crntRank = 6;

    double* mtxZ_d = NULL;
    double* mtxQ_trnc_d = NULL;

    bool debug = true;


    CHECK(hipMalloc((void**)&mtxZ_d, numOfRow * numOfClm * sizeof(double)));
    CHECK(hipMemcpy(mtxZ_d, mtxZ, numOfRow * numOfClm * sizeof(double), hipMemcpyHostToDevice));

    if(debug){
        printf("\n\n~~mtxZ~~\n\n");
        print_mtx_clm_d(mtxZ_d, numOfRow, numOfClm);
    }


    orth_QR(&mtxQ_trnc_d, mtxZ_d, numOfRow, numOfClm, crntRank);


    // printf("\n\n~~mtxY_Hat~~\n\n");
    // print_mtx_clm_d(mtxQ_trnc_d, numOfRow, crntRank);
    printf("\n\n~~Current Rarnk = %d~~\n\n", crntRank);

} // end of orth_QRtest1()

void orth_QRtest3()
{
// Define the dense matrixB column major
    double mtxZ[] = {
        1.1, 0.8, 3.0, 2.2, 1.4, 0.6, 7.7,
        2.2, 1.6, 4.1, 3.3, 2.5, 1.5, 5.6,
        3.3, 2.4, 5.2, 4.4, 3.6, 2.4, 9.6,
        4.4, 3.2, 6.3, 5.5, 4.7, 3.3, 8.8,
        5.5, 4.0, 7.4, 6.6, 5.8, 4.2, 7.0
    };

    int numOfRow = 7;
    int numOfClm = 5;
    int crntRank = 5;

    double* mtxZ_d = NULL;
    double* mtxQ_trnc_d = NULL;

    bool debug = true;


    CHECK(hipMalloc((void**)&mtxZ_d, numOfRow * numOfClm * sizeof(double)));
    CHECK(hipMemcpy(mtxZ_d, mtxZ, numOfRow * numOfClm * sizeof(double), hipMemcpyHostToDevice));

    if(debug){
        printf("\n\n~~mtxZ~~\n\n");
        print_mtx_clm_d(mtxZ_d, numOfRow, numOfClm);
    }


    orth_QR(&mtxQ_trnc_d, mtxZ_d, numOfRow, numOfClm, crntRank);

    // printf("\n\n~~mtxY_Hat~~\n\n");
    // print_mtx_clm_d(mtxQ_trnc_d, numOfRow, crntRank);
    printf("\n\n~~Current Rarnk = %d~~\n\n", crntRank);

} // end of orth_QRtest1()

void orth_QRtest4()
{
 // Define the dense matrixB column major
    double mtxZ[] = {
        1.2, 0.9, 3.1, 2.3, 1.5, 0.7, 2.1, 3.3, 0.1, 0.2, 0.3, 
        2.3, 1.8, 4.2, 3.4, 2.6, 1.6, 3.0, 4.4, 0.5, 0.7, 0.2, 
        3.4, 2.7, 5.3, 4.5, 3.7, 2.5, 4.9, 5.5, 0.3, 0.4, 0.5, 
        4.5, 3.6, 6.4, 5.6, 4.8, 3.4, 5.8, 6.6, 1.1, 1.2, 1.3, 
        5.6, 4.5, 7.5, 6.7, 5.9, 4.3, 6.7, 7.7, 1.9, 1.5, 1.8, 
        6.7, 5.4, 8.6, 7.8, 6.0, 5.2, 7.6, 8.8, 2.2, 2.3, 2.5, 
        7.8, 6.3, 9.7, 8.9, 7.1, 6.1, 8.5, 9.9, 2.9, 3.1, 3.2
    };


    int numOfRow = 11;
    int numOfClm = 7;
    int crntRank = 7;

    double* mtxZ_d = NULL;
    double* mtxQ_trnc_d = NULL;

    bool debug = true;


    CHECK(hipMalloc((void**)&mtxZ_d, numOfRow * numOfClm * sizeof(double)));
    CHECK(hipMemcpy(mtxZ_d, mtxZ, numOfRow * numOfClm * sizeof(double), hipMemcpyHostToDevice));

    if(debug){
        printf("\n\n~~mtxZ~~\n\n");
        print_mtx_clm_d(mtxZ_d, numOfRow, numOfClm);
    }


    orth_QR(&mtxQ_trnc_d, mtxZ_d, numOfRow, numOfClm, crntRank);

    // printf("\n\n~~mtxY_Hat~~\n\n");
    // print_mtx_clm_d(mtxQ_trnc_d, numOfRow, crntRank);
    printf("\n\n~~Current Rarnk = %d~~\n\n", crntRank);


} // end of orth_QRtest1()

void orth_QRtest5()
{
    // Define the dense matrixB column major
    // Define the dense matrixB column major
    double mtxZ[] = {
        1.2, 0.9, 3.1, 2.3, 1.5, 0.7, 2.1, 3.3, 2.5, 1.9, 0.9, 1.0, 1.1, 1.2, 1.3, 7.7, 6.4, 8.6, 8.8, 6.0,
        2.3, 1.8, 4.2, 3.4, 2.6, 1.6, 3.0, 4.4, 3.6, 2.8, 1.8, 2.1, 2.4, 2.6, 2.8, 6.2, 7.6, 8.8, 7.0, 6.4,
        3.4, 2.7, 5.3, 4.5, 3.7, 2.5, 4.9, 5.5, 4.7, 3.7, 2.7, 3.2, 3.7, 4.0, 4.3, 5.4, 6.5, 7.6, 8.2, 8.8,
        4.5, 3.6, 6.4, 5.6, 4.8, 3.4, 5.8, 6.6, 5.8, 4.6, 3.6, 4.3, 5.0, 5.4, 5.8, 8.8, 7.3, 9.7, 9.9, 7.1,
        5.6, 4.5, 7.5, 6.7, 5.9, 4.3, 6.7, 7.7, 6.9, 5.5, 4.5, 5.4, 6.3, 6.8, 7.3, 7.1, 8.5, 9.9, 8.1, 7.3,
        6.7, 5.4, 8.6, 7.8, 6.0, 5.2, 7.6, 8.8, 7.0, 6.4, 5.4, 6.5, 7.6, 8.2, 8.8, 6.3, 7.6, 8.9, 9.6, 10.3,
        7.8, 6.3, 9.7, 8.9, 7.1, 6.1, 8.5, 9.9, 8.1, 7.3, 6.3, 7.6, 8.9, 9.6, 10.3, 9.9, 8.2, 0.8, 9.0, 8.2,
        8.9, 7.2, 0.8, 9.0, 8.2, 7.0, 9.4, 0.1, 9.2, 8.2, 7.2, 8.7, 0.2, 0.5, 1.8, 8.0, 9.4, 0.3, 9.2, 8.2,
        9.0, 8.1, 1.9, 1.1, 9.3, 8.9, 0.3, 1.1, 0.3, 9.1, 8.1, 9.8, 1.5, 1.4, 2.4, 7.2, 8.7, 0.2, 0.4, 1.8,
        1.1, 9.0, 2.0, 2.2, 0.4, 9.8, 1.2, 2.2, 1.4, 0.2, 9.0, 0.9, 2.8, 2.8, 3.0, 1.0, 9.1, 1.9, 2.2, 0.4,
        2.2, 1.9, 3.1, 3.3, 1.5, 1.7, 2.1, 3.3, 2.5, 1.9, 0.9, 1.0, 1.1, 1.2, 1.3, 9.9, 1.2, 2.2, 1.4, 0.7,
        3.3, 2.8, 4.2, 4.4, 2.6, 2.6, 3.0, 4.4, 3.6, 2.8, 1.8, 2.1, 2.4, 2.6, 2.8, 9.0, 0.9, 2.8, 2.8, 3.0,
        4.4, 3.7, 5.3, 5.5, 3.7, 3.5, 4.9, 5.5, 4.7, 3.7, 2.7, 3.2, 3.7, 4.0, 4.3, 2.1, 0.7, 2.0, 3.2, 1.5,
        5.5, 4.6, 6.4, 6.6, 4.8, 4.4, 5.8, 6.6, 5.8, 4.6, 3.6, 4.3, 5.0, 5.4, 5.8, 0.8, 2.1, 3.2, 2.3, 1.0,
        6.6, 5.5, 7.5, 7.7, 5.9, 5.3, 6.7, 7.7, 6.9, 5.5, 4.5, 5.4, 6.3, 6.8, 7.3, 0.9, 1.0, 1.1, 1.2, 1.3
    };
    int numOfRow = 20;
    int numOfClm = 15;
    int crntRank = 15;

    double* mtxZ_d = NULL;
    double* mtxQ_trnc_d = NULL;

    bool debug = true;


    CHECK(hipMalloc((void**)&mtxZ_d, numOfRow * numOfClm * sizeof(double)));
    CHECK(hipMemcpy(mtxZ_d, mtxZ, numOfRow * numOfClm * sizeof(double), hipMemcpyHostToDevice));

    if(debug){
        printf("\n\n~~mtxZ~~\n\n");
        print_mtx_clm_d(mtxZ_d, numOfRow, numOfClm);
    }


    orth_QR(&mtxQ_trnc_d, mtxZ_d, numOfRow, numOfClm, crntRank);


    // printf("\n\n~~mtxY_Hat~~\n\n");
    // print_mtx_clm_d(mtxQ_trnc_d, numOfRow, crntRank);
    printf("\n\n~~Current Rarnk = %d~~\n\n", crntRank);

} // end of orth_QRtest1()